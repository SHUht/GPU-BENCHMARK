
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 1000000

__global__ void vectorAdd(float* A, float* B, float* C) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

void benchmarkVectorAdd() {
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    size_t size = N * sizeof(float);

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    auto start = std::chrono::high_resolution_clock::now();
    vectorAdd<<<numBlocks, blockSize>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::chrono::duration<double> duration = end - start;
    std::cout << "Vector Add Time: " << duration.count() << " seconds" << std::endl;

    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    benchmarkVectorAdd();
    return 0;
}